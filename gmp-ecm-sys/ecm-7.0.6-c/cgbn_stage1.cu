#include "hip/hip_runtime.h"
/* cgbn_stage1.h: header for CGBN (GPU) based ecm stage 1.

Copyright 2021 Seth Troisi

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 3 of the License, or (at your
option) any later version.

This program is distributed in the hope that it will be useful, but
WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for
more details.

You should have received a copy of the GNU General Public License
along with this program; see the file COPYING.  If not, see
http://www.gnu.org/licenses/ or write to the Free Software Foundation, Inc.,
51 Franklin St, Fifth Floor, Boston, MA 02110-1301, USA.
*/

#ifndef _CGBN_STAGE1_CU
#define _CGBN_STAGE1_CU 1

#ifndef __HIPCC__
#error "This file should only be compiled with nvcc"
#endif

#include "cgbn_stage1.h"

#include <cassert>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

// GMP import must proceed cgbn.h
#include <gmp.h>
#include <cgbn.h>
#include <hip/hip_runtime.h>

#include "cudacommon.h"

#include "ecm.h"
#include "ecm-gpu.h"


// See cgbn_error_t enum (cgbn.h:39)
#define cgbn_normalized_error ((cgbn_error_t) 14)
#define cgbn_positive_overflow ((cgbn_error_t) 15)
#define cgbn_negative_overflow ((cgbn_error_t) 16)

// Seems to adds very small overhead (1-10%)
#define VERIFY_NORMALIZED 0
// Adds even less overhead (<1%)
#define CHECK_ERROR 1

// Tested with check_gpuecm.sage
#define CARRY_BITS 6

// Can dramatically change compile time
#if 1
    #define FORCE_INLINE __forceinline__
#else
    #define FORCE_INLINE
#endif

// support routine copied from  "CGBN/samples/utility/support.h"
void cgbn_check(cgbn_error_report_t *report, const char *file=NULL, int32_t line=0) {
  // check for cgbn errors

  if(cgbn_error_report_check(report)) {
    fprintf (stderr, "\n");
    fprintf (stderr, "CGBN error occurred: %s\n", cgbn_error_string(report));

    if(report->_instance!=0xFFFFFFFF) {
      fprintf (stderr, "Error reported by instance %d", report->_instance);
      if(report->_blockIdx.x!=0xFFFFFFFF)
        fprintf (stderr, ", blockIdx=(%d, %d, %d)", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      if(report->_threadIdx.x!=0xFFFFFFFF)
        fprintf (stderr, ", threadIdx=(%d, %d, %d)", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
      fprintf (stderr, "\n");
    }
    else {
      fprintf (stderr, "Error reported by blockIdx=(%d %d %d)", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      fprintf (stderr, "threadIdx=(%d %d %d)\n", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
    }
    if(file!=NULL)
      fprintf (stderr, "file %s, line %d\n", file, line);
    exit(1);
  }
}

#define CGBN_CHECK(report) cgbn_check(report, __FILE__, __LINE__)

static
void to_mpz(mpz_t r, const uint32_t *x, uint32_t count) {
  mpz_import (r, count, -1, sizeof(uint32_t), 0, 0, x);
}

static
void from_mpz(const mpz_t s, uint32_t *x, uint32_t count) {
  size_t words;

  if(mpz_sizeinbase (s, 2) > count * 32) {
    fprintf (stderr, "from_mpz failed -- result does not fit\n");
    exit(EXIT_FAILURE);
  }

  mpz_export (x, &words, -1, sizeof(uint32_t), 0, 0, s);
  while(words<count)
    x[words++]=0;
}



// ---------------------------------------------------------------- //

// The CGBN context uses the following three parameters:
//   TBP             - threads per block (zero means to use the blockDim.x)
//   MAX_ROTATION    - must be small power of 2, imperically, 4 works well
//   CONSTANT_TIME   - require constant time algorithms (currently, constant time algorithms are not available)

// Locally it will also be helpful to have several parameters:
//   TPI             - threads per instance
//   BITS            - number of bits per instance

/* TODO test how this changes gpu_throughput_test */
/* NOTE: >= 512 may not be supported for > 2048 bit kernels */
const uint32_t TPB_DEFAULT = 256;

template<uint32_t tpi, uint32_t bits>
class cgbn_params_t {
  public:
  // parameters used by the CGBN context
  static const uint32_t TPB=TPB_DEFAULT;           // Reasonable default
  static const uint32_t MAX_ROTATION=4;            // good default value
  static const uint32_t SHM_LIMIT=0;               // no shared mem available
#ifndef _MSC_VER
  static const bool     CONSTANT_TIME=false;       // not implemented
#endif

  // parameters used locally in the application
  static const uint32_t TPI=tpi;                   // threads per instance
  static const uint32_t BITS=bits;                 // instance size
};


template<class params>
class curve_t {
  public:

  typedef cgbn_context_t<params::TPI, params>   context_t;
  typedef cgbn_env_t<context_t, params::BITS>   env_t;
  typedef typename env_t::cgbn_t                bn_t;
  typedef cgbn_mem_t<params::BITS>              mem_t;

  context_t _context;
  env_t     _env;
  int32_t   _instance; // which curve instance is this

  // Constructor
  __device__ FORCE_INLINE curve_t(cgbn_monitor_t monitor, cgbn_error_report_t *report, int32_t instance) :
      _context(monitor, report, (uint32_t)instance), _env(_context), _instance(instance) {}

  // Verify 0 <= r < modulus
  __device__ FORCE_INLINE void assert_normalized(bn_t &r, const bn_t &modulus) {
    //if (VERIFY_NORMALIZED && _context.check_errors())
    if (VERIFY_NORMALIZED && CHECK_ERROR) {

        // Negative overflow
        if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
            _context.report_error(cgbn_negative_overflow);
        }
        // Positive overflow
        if (cgbn_compare(_env, r, modulus) >= 0) {
            _context.report_error(cgbn_positive_overflow);
        }
    }
  }

  // Normalize after addition
  __device__ FORCE_INLINE void normalize_addition(bn_t &r, const bn_t &modulus) {
      if (cgbn_compare(_env, r, modulus) >= 0) {
          cgbn_sub(_env, r, r, modulus);
      }
  }

  // Normalize after subtraction (handled instead by checking carry)
  /*
  __device__ FORCE_INLINE void normalize_subtraction(bn_t &r, const bn_t &modulus) {
      if (cgbn_extract_bits_ui32(_env, r, params::BITS-1, 1)) {
          cgbn_add(_env, r, r, modulus);
      }
  }
  */

  /**
   * Calculate (r * m) / 2^32 mod modulus
   *
   * This removes a factor of 2^32 which is not present in m.
   * Otherwise m (really d) needs to be passed as a bigint not a uint32
   */
  __device__ FORCE_INLINE void special_mult_ui32(bn_t &r, uint32_t m, const bn_t &modulus, uint32_t np0) {
    //uint32_t thread_i = (blockIdx.x*blockDim.x + threadIdx.x)%params::TPI;
    bn_t temp;

    uint32_t carry_t1 = cgbn_mul_ui32(_env, r, r, m);
    uint32_t t1_0 = cgbn_extract_bits_ui32(_env, r, 0, 32);
    uint32_t q = t1_0 * np0;
    uint32_t carry_t2 = cgbn_mul_ui32(_env, temp, modulus, q);

    // Can I call dshift_right(1) directly?
    cgbn_shift_right(_env, r, r, 32);
    cgbn_shift_right(_env, temp, temp, 32);
    // Add back overflow carry
    cgbn_insert_bits_ui32(_env, r, r, params::BITS-32, 32, carry_t1);
    cgbn_insert_bits_ui32(_env, temp, temp, params::BITS-32, 32, carry_t2);

    if (VERIFY_NORMALIZED) {
        // (uint32 * X) >> 32 is always less than X
        assert_normalized(r, modulus);
        assert_normalized(temp, modulus);
    }

    // Can't overflow because of CARRY_BITS
    int32_t carry_q = cgbn_add(_env, r, r, temp);
    carry_q += cgbn_add_ui32(_env, r, r, t1_0 != 0); // add 1

    if (carry_q > 0) {
        // This should never happen,
        // if CHECK_ERROR, no need for the conditional call to cgbn_sub
        if (CHECK_ERROR) {
            _context.report_error(cgbn_positive_overflow);
        } else {
            cgbn_sub(_env, r, r, modulus);
        }
    }

    // 0 <= r, temp < modulus => r + temp + 1 < 2*modulus
    if (cgbn_compare(_env, r, modulus) >= 0) {
        cgbn_sub(_env, r, r, modulus);
    }
  }


  /**
   * Compute simultaneously
   * (q : u) <- [2](q : u)
   * (w : v) <- (q : u) + (w : v)
   * A second implementation previously existed in cudakernel_default.cu
   */
  __device__ FORCE_INLINE void double_add_v2(
          bn_t &q, bn_t &u,
          bn_t &w, bn_t &v,
          uint32_t d,
          const bn_t &modulus,
          const uint32_t np0) {
    // q = xA = aX
    // u = zA = aZ
    // w = xB = bX
    // v = zB = bZ

    /* Doesn't seem to be a large cost to using many extra variables */
    bn_t t, CB, DA, AA, BB, K, dK;

    /* Can maybe use one more bit if cgbn_add subtracts when carry happens */
    /* Might be nice to add a macro that verifies no carry out of cgbn_add */

    // Is there anything interesting like only one of these can overflow?
    cgbn_add(_env, t, v, w); // t = (bZ + bX)
    normalize_addition(t, modulus);
    if (cgbn_sub(_env, v, v, w)) // v = (bZ - bX)
        cgbn_add(_env, v, v, modulus);


    cgbn_add(_env, w, u, q); // w = (aZ + aX)
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, u, u, q)) // u = (aZ - aX)
        cgbn_add(_env, u, u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(t, modulus);
        assert_normalized(v, modulus);
        assert_normalized(w, modulus);
        assert_normalized(u, modulus);
    }

    cgbn_mont_mul(_env, CB, t, u, modulus, np0); // C*B
        normalize_addition(CB, modulus);
    cgbn_mont_mul(_env, DA, v, w, modulus, np0); // D*A
        normalize_addition(DA, modulus);

    /* Roughly 40% of time is spent in these two calls */
    cgbn_mont_sqr(_env, AA, w, modulus, np0);    // AA
    cgbn_mont_sqr(_env, BB, u, modulus, np0);    // BB
    normalize_addition(AA, modulus);
    normalize_addition(BB, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(CB, modulus);
        assert_normalized(DA, modulus);
        assert_normalized(AA, modulus);
        assert_normalized(BB, modulus);
    }

    // q = aX is finalized
    cgbn_mont_mul(_env, q, AA, BB, modulus, np0); // AA*BB
    normalize_addition(q, modulus);
        assert_normalized(q, modulus);

    if (cgbn_sub(_env, K, AA, BB)) // K = AA-BB
        cgbn_add(_env, K, K, modulus);

    // By definition of d = (sigma / 2^32) % MODN
    // K = k*R
    // dK = d*k*R = (K * R * sigma) >> 32
    cgbn_set(_env, dK, K);
    special_mult_ui32(dK, d, modulus, np0); // dK = K*d
        assert_normalized(dK, modulus);

    cgbn_add(_env, u, BB, dK); // BB + dK
    normalize_addition(u, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(K, modulus);
        assert_normalized(dK, modulus);
        assert_normalized(u, modulus);
    }

    // u = aZ is finalized
    cgbn_mont_mul(_env, u, K, u, modulus, np0); // K(BB+dK)
    normalize_addition(u, modulus);
        assert_normalized(u, modulus);

    cgbn_add(_env, w, DA, CB); // DA + CB
    normalize_addition(w, modulus);
    if (cgbn_sub(_env, v, DA, CB)) // DA - CB
        cgbn_add(_env, v, v, modulus);
    if (VERIFY_NORMALIZED) {
        assert_normalized(w, modulus);
        assert_normalized(v, modulus);
    }

    // w = bX is finalized
    cgbn_mont_sqr(_env, w, w, modulus, np0); // (DA+CB)^2 mod N
    normalize_addition(w, modulus);
        assert_normalized(w, modulus);

    cgbn_mont_sqr(_env, v, v, modulus, np0); // (DA-CB)^2 mod N
    normalize_addition(v, modulus);
        assert_normalized(v, modulus);

    // v = bZ is finalized
    cgbn_shift_left(_env, v, v, 1); // double
    normalize_addition(v, modulus);
        assert_normalized(v, modulus);
  }
};


// kernel implementation using cgbn
template<class params>
__global__ void kernel_double_add(
        cgbn_error_report_t *report,
        uint64_t s_bits,
        uint64_t s_bits_start,
        uint64_t s_bits_interval,
        uint32_t *gpu_s_bits,
        uint32_t *data,
        uint32_t count,
        uint32_t sigma_0,
        uint32_t np0
        ) {
  // decode an instance_i number from the blockIdx and threadIdx
  int32_t instance_i = (blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  if(instance_i >= count)
    return;

  /* Cast uint32_t array to mem_t */
  typename curve_t<params>::mem_t *data_cast = (typename curve_t<params>::mem_t*) data;

  cgbn_monitor_t monitor = CHECK_ERROR ? cgbn_report_monitor : cgbn_no_checks;

  curve_t<params> curve(monitor, report, instance_i);
  typename curve_t<params>::bn_t  aX, aZ, bX, bZ, modulus;

  { // Setup
      cgbn_load(curve._env, modulus, &data_cast[5*instance_i+0]);
      cgbn_load(curve._env, aX, &data_cast[5*instance_i+1]);
      cgbn_load(curve._env, aZ, &data_cast[5*instance_i+2]);
      cgbn_load(curve._env, bX, &data_cast[5*instance_i+3]);
      cgbn_load(curve._env, bZ, &data_cast[5*instance_i+4]);

      /* Convert points to mont, has a miniscule bit of overhead with batching. */
      uint32_t np0_test = cgbn_bn2mont(curve._env, aX, aX, modulus);
      assert(np0 == np0_test);

      cgbn_bn2mont(curve._env, aZ, aZ, modulus);
      cgbn_bn2mont(curve._env, bX, bX, modulus);
      cgbn_bn2mont(curve._env, bZ, bZ, modulus);

      {
        curve.assert_normalized(aX, modulus);
        curve.assert_normalized(aZ, modulus);
        curve.assert_normalized(bX, modulus);
        curve.assert_normalized(bZ, modulus);
      }
  }

  /* Initially
     P_a = (aX, aZ) contains P
     P_b = (bX, bZ) contains 2P */

  uint32_t d = sigma_0 + instance_i;
  int swapped = 0;
  for (uint64_t b = s_bits_start; b < s_bits_start + s_bits_interval; b++) {
    /* Process bits from MSB to LSB, last index to first index
     * b counts from 0 to s_num_bits */
    uint64_t nth = s_bits - 1 - b;
    int bit = (gpu_s_bits[nth/32] >> (nth&31)) & 1;
    if (bit != swapped) {
        swapped = !swapped;
        cgbn_swap(curve._env, aX, bX);
        cgbn_swap(curve._env, aZ, bZ);
    }
    curve.double_add_v2(aX, aZ, bX, bZ, d, modulus, np0);
  }

  if (swapped) {
    cgbn_swap(curve._env, aX, bX);
    cgbn_swap(curve._env, aZ, bZ);
  }

  { // Final output
    // Convert everything back to bn
    cgbn_mont2bn(curve._env, aX, aX, modulus, np0);
    cgbn_mont2bn(curve._env, aZ, aZ, modulus, np0);
    cgbn_mont2bn(curve._env, bX, bX, modulus, np0);
    cgbn_mont2bn(curve._env, bZ, bZ, modulus, np0);

    {
      curve.assert_normalized(aX, modulus);
      curve.assert_normalized(aZ, modulus);
      curve.assert_normalized(bX, modulus);
      curve.assert_normalized(bZ, modulus);
    }
    cgbn_store(curve._env, &data_cast[5*instance_i+1], aX);
    cgbn_store(curve._env, &data_cast[5*instance_i+2], aZ);
    cgbn_store(curve._env, &data_cast[5*instance_i+3], bX);
    cgbn_store(curve._env, &data_cast[5*instance_i+4], bZ);
  }
}


static
int findfactor(mpz_t factor, const mpz_t N, const mpz_t x_final, const mpz_t z_final) {
    // XXX: combine / refactor logic with cudawrapper.c findfactor

    /* Check if factor found */
    bool inverted = mpz_invert(factor, z_final, N);    // aZ ^ (N-2) % N

    if (inverted) {
        mpz_mul(factor, x_final, factor);         // aX * aZ^-1
        mpz_mod(factor, factor, N);             // "Residual"
        return ECM_NO_FACTOR_FOUND;
    }

    mpz_gcd(factor, z_final, N);
    return ECM_FACTOR_FOUND_STEP1;
}


static
int verify_size_of_n(const mpz_t N, size_t max_bits) {
  size_t n_log2 = mpz_sizeinbase(N, 2);

  /* Using check_gpuecm.sage it looks like 4 bits would suffice. */
  size_t max_usable_bits = max_bits - CARRY_BITS;

  if (n_log2 <= max_usable_bits)
    return ECM_NO_FACTOR_FOUND;

  outputf (OUTPUT_ERROR, "GPU: N(%d bits) + carry(%d bits) > BITS(%d)\n",
      n_log2, CARRY_BITS, max_bits);
  outputf (OUTPUT_ERROR, "GPU: Error, input number should be stricly lower than 2^%d\n",
      max_usable_bits);
  return ECM_ERROR;
}


static
uint32_t find_np0(const mpz_t N) {
  uint32_t np0;
  mpz_t temp;
  mpz_init(temp);
  mpz_ui_pow_ui(temp, 2, 32);
  assert(mpz_invert(temp, N, temp));
  np0 = -mpz_get_ui(temp);
  mpz_clear(temp);
  return np0;
}


static
uint32_t* allocate_and_set_s_bits(const mpz_t s, uint64_t *nbits) {
  uint64_t num_bits = *nbits = mpz_sizeinbase (s, 2);

  uint64_t allocated = (num_bits + 31) / 32;
  uint32_t *s_bits = (uint32_t*) malloc (sizeof(uint32_t) * allocated);

  uint64_t countp;
  mpz_export (s_bits, &countp, -1, sizeof(uint32_t), 0, 0, s);
  assert (countp == allocated);

  return s_bits;
}


static
uint32_t* set_p_2p(const mpz_t N,
                   uint32_t curves, uint32_t sigma,
                   uint32_t BITS, size_t *data_size) {
  /**
   * Store 5 numbers per curve:
   * N, P_a (x, z), P_b (x, z)
   *
   * P_a is initialized with (2, 1)
   * P_b (for the doubled terms) is initialized with (9, 64 * d + 8)
   */

  const size_t limbs_per = BITS/32;
  *data_size = 5 * curves * limbs_per * sizeof(uint32_t);
  uint32_t *data = (uint32_t*) malloc(*data_size);
  uint32_t *datum = data;

  mpz_t x;
  mpz_init(x);
  for(int index = 0; index < curves; index++) {
      // d = (sigma / 2^32) mod N BUT 2^32 handled by special_mul_ui32
      uint32_t d = sigma + index;

      // Modulo (N)
      from_mpz(N, datum + 0 * limbs_per, BITS/32);

      // P1 (X, Z)
      mpz_set_ui(x, 2);
      from_mpz(x, datum + 1 * limbs_per, BITS/32);
      mpz_set_ui(x, 1);
      from_mpz(x, datum + 2 * limbs_per, BITS/32);

      // 2P = P2 (X, Z)
      // P2_x = 9
      mpz_set_ui(x, 9);
      from_mpz(x, datum + 3 * limbs_per, BITS/32);

      // d = sigma * mod_inverse(2 ** 32, N)
      mpz_ui_pow_ui(x, 2, 32);
      mpz_invert(x, x, N);
      mpz_mul_ui(x, x, d);
      // P2_x = 64 * d + 8;
      mpz_mul_ui(x, x, 64);
      mpz_add_ui(x, x, 8);
      mpz_mod(x, x, N);

      outputf (OUTPUT_TRACE, "sigma %d => P2_y: %Zd\n", d, x);
      from_mpz(x, datum + 4 * limbs_per, BITS/32);
      datum += 5 * limbs_per;
  }
  mpz_clear(x);
  return data;
}


static
int process_results(mpz_t *factors, int *array_found,
                    const mpz_t N,
                    const uint32_t *data, uint32_t cgbn_bits,
                    int curves, uint32_t sigma) {
  mpz_t x_final, z_final, modulo;
  mpz_init(modulo);
  mpz_init(x_final);
  mpz_init(z_final);

  const uint32_t limbs_per = cgbn_bits / 32;

  int youpi = ECM_NO_FACTOR_FOUND;
  int errors = 0;
  for(size_t i = 0; i < curves; i++) {
    const uint32_t *datum = data + (5 * i * limbs_per);;

    if (test_verbose (OUTPUT_TRACE) && i == 0) {
      to_mpz(modulo, datum + 0 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 modulo: %Zd\n", modulo);

      to_mpz(x_final, datum + 1 * limbs_per, limbs_per);
      to_mpz(z_final, datum + 2 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 pA: (%Zd, %Zd)\n", x_final, z_final);

      to_mpz(x_final, datum + 3 * limbs_per, limbs_per);
      to_mpz(z_final, datum + 4 * limbs_per, limbs_per);
      outputf (OUTPUT_TRACE, "index: 0 pB: (%Zd, %Zd)\n", x_final, z_final);
    }

    // Make sure we were testing the right number.
    to_mpz(modulo, datum + 0 * limbs_per, limbs_per);
    assert(mpz_cmp(modulo, N) == 0);

    to_mpz(x_final, datum + 1 * limbs_per, limbs_per);
    to_mpz(z_final, datum + 2 * limbs_per, limbs_per);

    /* Very suspicious for (x_final, z_final) to match (x_0, z_0) == (2, 1)
     * Can happen when
     * 1. block calculation performed incorrectly (and some blocks not run)
     * 2. Kernel didn't run because not enough register
     * 3. nvcc links old version of kernel when something changed
     */
    if (mpz_cmp_ui (x_final, 2) == 0 && mpz_cmp_ui (z_final, 1) == 0) {
      errors += 1;
      if (errors < 10 || errors % 100 == 1)
        outputf (OUTPUT_ERROR, "GPU: curve %d didn't compute?\n", i);
    }

    array_found[i] = findfactor(factors[i], N, x_final, z_final);
    if (array_found[i] != ECM_NO_FACTOR_FOUND) {
      youpi = array_found[i];
      outputf (OUTPUT_NORMAL, "GPU: factor %Zd found in Step 1 with curve %ld (-sigma %d:%lu)\n",
          factors[i], i, ECM_PARAM_BATCH_32BITS_D, sigma + i);
    }
  }

  mpz_clear(modulo);
  mpz_clear(x_final);
  mpz_clear(z_final);

#ifdef IS_DEV_BUILD
  if (errors)
        outputf (OUTPUT_ERROR, "Had %d errors. Try `make clean; make` or reducing TPB_DEFAULT\n",
            errors);
#endif

  if (errors > 2)
      return ECM_ERROR;

  return youpi;
}


static
int print_nth_batch(int n)
{
  return ((n < 3) ||
          (n < 30 && n % 10 == 0) ||
          (n < 500 && n % 100 == 0) ||
          (n < 5000 && n % 1000 == 0) ||
          (n % 10000 == 0));
}

int cgbn_ecm_stage1(mpz_t *factors, int *array_found,
             const mpz_t N, const mpz_t s,
             uint32_t curves, uint32_t sigma,
             float *gputime, int verbose)
{
  assert( sigma > 0 );
  assert( ((uint64_t) sigma + curves) <= 0xFFFFFFFF ); // no overflow

  uint64_t s_num_bits;
  uint32_t *s_bits = allocate_and_set_s_bits(s, &s_num_bits);
  if (s_num_bits >= 4000000000)
      outputf (OUTPUT_ALWAYS, "GPU: Very Large B1! Check magnitute of B1.\n");

  if (s_num_bits >= 100000000)
      outputf (OUTPUT_NORMAL, "GPU: Large B1, S = %'lu bits = %d MB\n",
               s_num_bits, s_num_bits >> 23);
  assert( s_bits != NULL );

  hipEvent_t global_start, batch_start, stop;
  CUDA_CHECK(hipEventCreate (&global_start));
  CUDA_CHECK(hipEventCreate (&batch_start));
  CUDA_CHECK(hipEventCreate (&stop));
  CUDA_CHECK(hipEventRecord (global_start));

  // Copy s_bits
  uint32_t *gpu_s_bits;
  uint32_t s_words = (s_num_bits + 31) / 32;
  CUDA_CHECK(hipMalloc((void **)&gpu_s_bits, sizeof(uint32_t) * s_words));
  CUDA_CHECK(hipMemcpy(gpu_s_bits, s_bits, sizeof(uint32_t) * s_words, hipMemcpyHostToDevice));

  cgbn_error_report_t *report;
  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  size_t    data_size;
  uint32_t *data, *gpu_data;

  uint32_t  BITS = 0;        // kernel bits
  int32_t   TPB=TPB_DEFAULT; // Always the same default
  int32_t   TPI;
  int32_t   IPB;             // IPB = TPB / TPI, instances per block
  size_t    BLOCK_COUNT;     // How many blocks to cover all curves

  /**
   * Smaller TPI is faster, Larger TPI is needed for large inputs.
   * N > 512 TPI=8 | N > 2048 TPI=16 | N > 8192 TPI=32
   *
   * Larger takes longer to compile (and increases binary size)
   * No GPU, No CGBN | ecm 3.4M, 2 seconds to compile
   * GPU, No CGBN    | ecm 3.5M, 3 seconds
   * (8, 1024)       | ecm 3.8M, 12 seconds
   * (16,8192)       | ecm 4.2M, 1 minute
   * (32,16384)      | ecm 4.2M, 1 minute
   * (32,32768)      | ecm 5.2M, 4.7 minutes
   */
  /* NOTE: Custom kernel changes here
   * For "Compiling custom kernel for %d bits should be XX% faster"
   * Change the 512 in cgbn_params_t<4, 512> cgbn_params_small;
   * to the suggested value (a multiple of 32 >= bits + 6).
   * You may need to change the 4 to an 8 (or 16) if bits >512, >2048
   */
  /** TODO: try with const vector for BITs/TPI, see if compiler is happy */
  std::vector<uint32_t> available_kernels;

  typedef cgbn_params_t<4, 512>   cgbn_params_small;
  typedef cgbn_params_t<8, 1024>  cgbn_params_medium;
  available_kernels.push_back((uint32_t)cgbn_params_small::BITS);
  available_kernels.push_back((uint32_t)cgbn_params_medium::BITS);

#ifndef IS_DEV_BUILD
  /**
   * TPI and BITS have to be set at compile time. Adding multiple cgbn_params
   * (and their associated kernels) allows for better dynamic selection based
   * on the size of N (e.g. N < 1024, N < 2048, N < 4096) but increase compile
   * time and binary size. A few reasonable sizes are included and a verbose
   * warning is printed when a particular N might benefit from a custom sized
   * kernel.
   */
  typedef cgbn_params_t<8, 1536>  cgbn_params_1536;
  typedef cgbn_params_t<8, 2048>  cgbn_params_2048;
  typedef cgbn_params_t<16, 3072> cgbn_params_3072;
  typedef cgbn_params_t<16, 4096> cgbn_params_4096;
  available_kernels.push_back((uint32_t)cgbn_params_1536::BITS);
  available_kernels.push_back((uint32_t)cgbn_params_2048::BITS);
  available_kernels.push_back((uint32_t)cgbn_params_3072::BITS);
  available_kernels.push_back((uint32_t)cgbn_params_4096::BITS);
#endif

  /* Pointer to CUDA kernel. */
  void(*kernel)(cgbn_error_report_t *, uint64_t, uint64_t, uint64_t,
        uint32_t*, uint32_t*, uint32_t, uint32_t, uint32_t) = NULL;

  size_t n_log2 = mpz_sizeinbase(N, 2);
  for (int k_i = 0; k_i < available_kernels.size(); k_i++) {
    uint32_t kernel_bits = available_kernels[k_i];
    if (kernel_bits >= n_log2 + CARRY_BITS) {
      BITS = kernel_bits;
      assert( BITS % 32 == 0 );

      /* Print some debug info about kernel. */
      /* TODO: return kernelAttr and validate maxThreadsPerBlock. */
      if (BITS == cgbn_params_small::BITS) {
        TPI = cgbn_params_small::TPI;
        kernel = kernel_double_add<cgbn_params_small>;
      } else if (BITS == cgbn_params_medium::BITS) {
        TPI = cgbn_params_medium::TPI;
        kernel = kernel_double_add<cgbn_params_medium>;
#ifndef IS_DEV_BUILD
      } else if (BITS == cgbn_params_1536::BITS) {
        TPI = cgbn_params_1536::TPI;
        kernel = kernel_double_add<cgbn_params_1536>;
      } else if (BITS == cgbn_params_2048::BITS) {
        TPI = cgbn_params_2048::TPI;
        kernel = kernel_double_add<cgbn_params_2048>;
      } else if (BITS == cgbn_params_3072::BITS) {
        TPI = cgbn_params_3072::TPI;
        kernel = kernel_double_add<cgbn_params_3072>;
      } else if (BITS == cgbn_params_4096::BITS) {
        TPI = cgbn_params_4096::TPI;
        kernel = kernel_double_add<cgbn_params_4096>;
#endif
      } else {
        outputf (OUTPUT_ERROR, "CGBN kernel not found for %d bits\n", BITS);
        return ECM_ERROR;
      }

      IPB = TPB / TPI;
      BLOCK_COUNT = (curves + IPB - 1) / IPB;

      break;
    }
  }
  if (BITS == 0 || kernel == NULL)
    {
      outputf (OUTPUT_ERROR, "No available CGBN Kernel large enough to process N(%d bits)\n", n_log2);
      return ECM_ERROR;
    }

  kernel_info((const void*)kernel_double_add<cgbn_params_medium>, verbose);

  /* Alert that recompiling with a smaller kernel would likely improve speed */
  {
    size_t optimized_bits = ((n_log2 + CARRY_BITS + 127)/128) * 128;
    /* Assume speed is roughly O(N) but slightly slower for not being a power of two */
    float pct_faster = 90 * BITS / optimized_bits;

    if (pct_faster > 110) {
      outputf (OUTPUT_VERBOSE, "Compiling custom kernel for %d bits should be ~%.0f%% faster see README.gpu\n",
              optimized_bits, pct_faster);
    }
  }

  int youpi = verify_size_of_n(N, BITS);
  if (youpi != ECM_NO_FACTOR_FOUND) {
    return youpi;
  }

  /* Consistency check that struct cgbn_mem_t is byte aligned without extra fields. */
  assert( sizeof(curve_t<cgbn_params_small>::mem_t) == cgbn_params_small::BITS/8 );
  assert( sizeof(curve_t<cgbn_params_medium>::mem_t) == cgbn_params_medium::BITS/8 );
  data = set_p_2p(N, curves, sigma, BITS, &data_size);

  /* np0 is -(N^-1 mod 2**32), used for montgomery representation */
  uint32_t np0 = find_np0(N);

  // Copy data
  outputf (OUTPUT_VERBOSE, "Copying %'lu bytes of curves data to GPU\n", data_size);
  CUDA_CHECK(hipMalloc((void **)&gpu_data, data_size));
  CUDA_CHECK(hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice));

  outputf (OUTPUT_VERBOSE,
          "CGBN<%d, %d> running kernel<%d block x %d threads> input number is %d bits\n",
          BITS, TPI, BLOCK_COUNT, TPB, n_log2);

  /* First bit (doubling) is handled in set_p_2p */
  uint64_t s_partial = 1;

  /* Start with small batches and increase till timing is ~100ms */
  uint64_t batch_size = 200;

  int batches_complete = 0;
  /* gputime and batch_time are measured in ms */
  float batch_time = 0;

  while (s_partial < s_num_bits) {
    /* decrease batch_size for final batch if needed */
    batch_size = std::min(s_num_bits - s_partial, batch_size);

    /* print ETA with lessing frequently, 5 early + 5 per 10s + 5 per 100s + every 1000s */
    if (print_nth_batch (batches_complete)) {
      outputf (OUTPUT_VERBOSE, "Computing %d bits/call, %lu/%lu (%.1f%%)",
          batch_size, s_partial, s_num_bits, 100.0 * s_partial / s_num_bits);
      if (batches_complete < 2 || *gputime < 1000) {
        outputf (OUTPUT_VERBOSE, "\n");
      } else {
        float estimated_total = (*gputime) * ((float) s_num_bits) / s_partial;
        float eta = estimated_total - (*gputime);
        outputf (OUTPUT_VERBOSE, ", ETA %.f + %.f = %.f seconds (~%.f ms/curves)\n",
                eta / 1000, *gputime / 1000, estimated_total / 1000,
                estimated_total / curves);
      }
    }

    CUDA_CHECK(hipEventRecord (batch_start));

    /* Call CUDA Kernel. */
    assert (kernel != NULL);
    (*kernel)<<<BLOCK_COUNT, TPB>>>(report, s_num_bits, s_partial, batch_size, gpu_s_bits, gpu_data, curves, sigma, np0);

    s_partial += batch_size;
    batches_complete++;

    /* error report uses managed memory, sync the device and check for cgbn errors */
    CUDA_CHECK(hipDeviceSynchronize());
    if (report->_error)
      outputf (OUTPUT_ERROR, "\n\nerror: %d\n", report->_error);
    CGBN_CHECK(report);

    CUDA_CHECK(hipEventRecord (stop));
    CUDA_CHECK(hipEventSynchronize (stop));
    hipEventElapsedTime (&batch_time, batch_start, stop);
    hipEventElapsedTime (gputime, global_start, stop);
    /* Adjust batch_size to aim for 100ms */
    if (batch_time < 80) {
      batch_size = 11*batch_size/10;
    } else if (batch_time > 120) {
      batch_size = max(100ul, 9*batch_size / 10);
    }
  }

  // Copy data back from GPU memory
  outputf (OUTPUT_VERBOSE, "Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(data, gpu_data, data_size, hipMemcpyDeviceToHost));

  hipEventElapsedTime (gputime, global_start, stop);

  youpi = process_results(factors, array_found, N, data, BITS, curves, sigma);

  // clean up
  CUDA_CHECK(hipFree(gpu_s_bits));
  CUDA_CHECK(hipFree(gpu_data));
  CUDA_CHECK(cgbn_error_report_free(report));
  CUDA_CHECK(hipEventDestroy (global_start));
  CUDA_CHECK(hipEventDestroy (batch_start));
  CUDA_CHECK(hipEventDestroy (stop));

  free(s_bits);
  free(data);

  return youpi;
}

#ifdef __CUDA_ARCH__
  #if __CUDA_ARCH__ < 350
    #error "Unsupported architecture"
  #endif
#endif

#endif  /* _CGBN_STAGE1_CU */
